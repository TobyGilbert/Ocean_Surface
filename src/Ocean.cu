#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------------------------------------------
/// @author Toby Gilbert
// ----------------------------------------------------------------------------------------------------------------------------------------
#include "Ocean.h"
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <glm/glm.hpp>
#include <complex>
#include <hiprand.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <surface_functions.h>
#include <hip/hip_vector_types.h>
/// @brief Given a time you can create a field of frequency amplitudes
/// @param d_h0Pointer An OpenGL buffer which stores a set of amplitudes and phases at time zero
/// @param d_htPointer An OpenGL buffer for outputting the frequency amplitude field
/// @param _time The current simulation time
/// @param _res The simulation resolution
// ----------------------------------------------------------------------------------------------------------------------------------------
__global__ void frequencyDomain(float2* d_h0Pointer, float2* d_htPointer, float _time, int _res){
    // A constant for the accelleration due to gravity
    const float g = 9.81;

    // A 2D vector to represent a position on the grid with constraits -(_res/2) <= k < (_res/2)
    float2 k;
    k.x = float((threadIdx.x - (_res * floor(double(threadIdx.x / _res)))) - (_res/2));
    k.y = float(((blockIdx.x * (blockDim.x/_res)) + ceil(double(threadIdx.x / _res))) - (_res/2));
    float kLen = sqrt(double(k.x*k.x + k.y*k.y));

    // Calculate the wave frequency
    float w = sqrt(double(g * kLen));

    // complexExp holds the complex exponential where the x value stores the real part and the y value stores the imaginary part
    float2 complexExp;
    complexExp.x = sin(w * _time);
    complexExp.y = cos(w * _time);

    float2 complexExpConjugate;
    complexExpConjugate.x = complexExp.x;
    complexExpConjugate.y = -complexExp.y;

    int blockNum =(( _res * _res )/ blockDim.x) - 1;

    float2 h0 = d_h0Pointer[(blockIdx.x * blockDim.x) + threadIdx.x];
    float2 h0conjugate = d_h0Pointer[((blockNum - blockIdx.x) * blockDim.x) + ((blockDim.x - 1) - threadIdx.x)];

    // Swap the imaginary parts sign
    h0conjugate.y = -h0conjugate.y;

    // Equation 26 of Tessendorf's paper h(k,t) = h0(k)exp{iw(k)t} + ~h0(-k)exp{-iw(k)t}
    float2 h;
    h.x = (h0.x * complexExp.x - h0.y * complexExp.y);
    h.y = (h0.x * complexExp.x + h0.y * complexExp.y);

    float2 hStar;
    hStar.x = (h0conjugate.x * complexExpConjugate.x - h0conjugate.y * complexExpConjugate.y) ;
    hStar.y = (h0conjugate.x * complexExpConjugate.x - h0conjugate.y * complexExpConjugate.y) ;

    // Output h(k,t) term to d_htPointer buffer which represents a set of points in the frequency domain
    float2 hTilde;
    hTilde.x= h.x + hStar.x;
    hTilde.y = h.y + hStar.y;

    d_htPointer[(blockIdx.x * blockDim.x) + threadIdx.x].x = hTilde.x;
    d_htPointer[(blockIdx.x * blockDim.x) + threadIdx.x].y = hTilde.y;
}
// ----------------------------------------------------------------------------------------------------------------------------------------
/// @brief Once inverse FFT has been performed points in the frequency domain are converted to the spatial domain
/// and can be used to update the heights
/// @param d_position An OpenGL buffer for storing the current positions of the vertices in the grid
/// @param d_height An OpenGL buffer which holds the new heights of grid positions
/// @param d_normal An OpenGL buffer which holds the normals
/// @param d_xDisplacement An OpenGL buffer for storing the displacment in the x axis
/// @param _res The resolution of the grid
/// @param _scale Scales the amplitude of the waves
// ----------------------------------------------------------------------------------------------------------------------------------------
__global__ void height(float3* d_position,  float2* d_height, float2* d_chopX, float2* d_chopZ, float _choppiness, int _res, float _scale){
    // A vertex on the grid
    int u = int(threadIdx.x - (_res * floor(double(threadIdx.x / _res))));
    int v = int((blockIdx.x * (blockDim.x/(float)_res)) + ceil(double(threadIdx.x / _res)));

    // Sign correction - Unsure why this is needed
    float sign = 1.0;
    if ((u+v) % 2 != 0){
        sign = -1.0;
    }

    // Update the heights of the vertices
    float prevX = d_position[(blockIdx.x * blockDim.x) + threadIdx.x].x;
    float prevZ = d_position[(blockIdx.x * blockDim.x) + threadIdx.x].z;
    float xDisp = _choppiness * (d_chopX[(blockIdx.x * blockDim.x) + threadIdx.x].x  /_scale) * sign;
    float zDisp = _choppiness * (d_chopZ[(blockIdx.x * blockDim.x) + threadIdx.x].x  /_scale) * sign;
    float height =  ((d_height[(blockIdx.x * blockDim.x) + threadIdx.x].x / _scale) * sign ) / 255.0f;
    float newX = prevX +xDisp;
    float newZ = prevZ + zDisp;

    d_position[(blockIdx.x * blockDim.x) + threadIdx.x].x = newX;
    d_position[(blockIdx.x * blockDim.x) + threadIdx.x].y =height;
    d_position[(blockIdx.x * blockDim.x) + threadIdx.x].z = newZ;
}

__global__ void calculateNormals(float3* d_position, float3* d_normals, int _res){

    float3 norm = make_float3(0.0, 0.0, 0.0);
    float3 posL, posR, posD, posU;
    /// @todo remove branching conditions
    if (((blockIdx.x * blockDim.x) + threadIdx.x) >= 1){
        posL = (d_position[((blockIdx.x * blockDim.x) + threadIdx.x) - 1]);
    }
    else{
        posL = (d_position[_res]); // A position on a neighbouring tile
    }
    if (((blockIdx.x * blockDim.x) + threadIdx.x) <=(_res*_res)-2){
        posR = (d_position[((blockIdx.x * blockDim.x) + threadIdx.x) + 1]);
    }
    else{
        posR = d_position[_res*_res - _res]; // A position on a neighbouring tile
    }
    if (((blockIdx.x * blockDim.x) + threadIdx.x) >= _res){
        posU = (d_position[((blockIdx.x * blockDim.x) + threadIdx.x) - _res]);
    }
    else{
        posU = d_position[_res*_res-_res + threadIdx.x];
    }
    if (((blockIdx.x * blockDim.x) + threadIdx.x) <= (_res*_res)-_res-1){
        posD = (d_position[((blockIdx.x * blockDim.x) + threadIdx.x) + _res]);
    }
    else{
        posD = d_position[threadIdx.x];
    }

    float3 leftVec, rightVec, topVec, bottomVec;
    float3 centerVec = d_position[((blockIdx.x * blockDim.x) + threadIdx.x)];
    leftVec =  posL - centerVec;
    leftVec.y *= 100.0;
    rightVec = posR - centerVec;
    rightVec.y *= 100.0;
    topVec = posU - centerVec;
    topVec.y *= 100.0;
    bottomVec =  posD - centerVec;
    bottomVec.y *= 100.0;

    float3 tmpNorm1 = normalize(cross(leftVec, topVec));
    float3 tmpNorm2 = normalize(cross(topVec, rightVec));
    float3 tmpNorm3 = normalize(cross(rightVec, bottomVec));
    float3 tmpNorm4 = normalize(cross(bottomVec, leftVec));

    tmpNorm1.y = fabs(tmpNorm1.y);
    tmpNorm2.y = fabs(tmpNorm2.y);
    tmpNorm3.y = fabs(tmpNorm3.y);
    tmpNorm4.y = fabs(tmpNorm4.y);
    norm = normalize((tmpNorm1 + tmpNorm2 + tmpNorm3 + tmpNorm4));

    // Update the normals buffer
    d_normals[(blockIdx.x * blockDim.x) + threadIdx.x] = norm;
}

// ----------------------------------------------------------------------------------------------------------------------------------------
/// @brief Create x displacement in in the frequency domain
/// @param
/// @param d_xDisplacement An OpenGL buffer to store the x displacement in the frequency domain
/// @param d_zDisplacement An OpenGL buffer to store the z displacement in the frequency domain
/// @param _res The resolution of the grid
// ----------------------------------------------------------------------------------------------------------------------------------------
__global__ void choppiness(float2* d_Ht, float2* d_chopX, float2* d_chopZ, float2 _windSpeed){
    // k - A position on the grid
    float2 k;
    k.x = _windSpeed.x;
    k.y = _windSpeed.y;

    float kLen = sqrt(double(k.x*k.x + k.y*k.y));

    float Kx = k.x / kLen;
    float Kz = k.y / kLen;

    if (kLen == 0.0){
        Kx = 0.0;
        Kz = 0.0;
    }

    d_chopX[(blockIdx.x * blockDim.x) + threadIdx.x].x = 0.0;
    d_chopX[(blockIdx.x * blockDim.x) + threadIdx.x].y = d_Ht[(blockIdx.x * blockDim.x) + threadIdx.x].y * -Kx;

    d_chopZ[(blockIdx.x * blockDim.x) + threadIdx.x].x = 0.0;
    d_chopZ[(blockIdx.x * blockDim.x) + threadIdx.x].y = d_Ht[(blockIdx.x * blockDim.x) + threadIdx.x].y * -Kz;
}
// ----------------------------------------------------------------------------------------------------------------------------------------
void updateFrequencyDomain(float2 *d_h0, float2 *d_ht, float _time, int _res){
    int numBlocks =( _res * _res )/ 1024;
    frequencyDomain<<<numBlocks, 1024>>>(d_h0, d_ht, _time, _res);
}
// ----------------------------------------------------------------------------------------------------------------------------------------
void updateHeight(float3* d_position, float3* d_norms, float2* d_height, float2* d_chopX, float2* d_chopZ, float _choppiness, int _res, float _scale){
    int numBlocks =( _res * _res )/ 1024;
    height<<<numBlocks, 1024>>>(d_position, d_height, d_chopX, d_chopZ, _choppiness,  _res, _scale);

    hipDeviceSynchronize();

    calculateNormals<<<numBlocks, 1024>>>(d_position, d_norms, _res);
}
// ----------------------------------------------------------------------------------------------------------------------------------------
void addChoppiness(float2* d_Heights, float2* d_chopX, float2* d_chopZ, int _res, float2 _windDirection){
    int numBlocks =( _res * _res )/ 1024;
    choppiness<<<numBlocks, 1024>>>(d_Heights, d_chopX, d_chopZ, _windDirection);
}
// ----------------------------------------------------------------------------------------------------------------------------------------
